#include "hip/hip_runtime.h"
// ---------------------------------------------------------
// Author: Andy Zeng, Princeton University, 2016
// ---------------------------------------------------------
#include "tsdf_fusion/tsdf_cuda.cuh"
#include "tsdf_fusion/utils.hpp"

#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>


// CUDA kernel function to integrate a TSDF voxel volume given depth images
__global__
void Integrate(float * cam_K, float * cam2base, float * depth_im,
               int im_height, int im_width, int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,
               float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z, float voxel_size, float trunc_margin,
               float * voxel_grid_TSDF, float * voxel_grid_weight) {

    int pt_grid_z = blockIdx.x;
    int pt_grid_y = threadIdx.x;

    for (int pt_grid_x = 0; pt_grid_x < voxel_grid_dim_x; ++pt_grid_x) {

        // Convert voxel center from grid coordinates to base frame camera coordinates
        float pt_base_x = voxel_grid_origin_x + pt_grid_x * voxel_size;
        float pt_base_y = voxel_grid_origin_y + pt_grid_y * voxel_size;
        float pt_base_z = voxel_grid_origin_z + pt_grid_z * voxel_size;

        // Convert from base frame camera coordinates to current frame camera coordinates
        float tmp_pt[3] = {0};
        tmp_pt[0] = pt_base_x - cam2base[0 * 4 + 3];
        tmp_pt[1] = pt_base_y - cam2base[1 * 4 + 3];
        tmp_pt[2] = pt_base_z - cam2base[2 * 4 + 3];
        float pt_cam_x = cam2base[0 * 4 + 0] * tmp_pt[0] + cam2base[1 * 4 + 0] * tmp_pt[1] + cam2base[2 * 4 + 0] * tmp_pt[2];
        float pt_cam_y = cam2base[0 * 4 + 1] * tmp_pt[0] + cam2base[1 * 4 + 1] * tmp_pt[1] + cam2base[2 * 4 + 1] * tmp_pt[2];
        float pt_cam_z = cam2base[0 * 4 + 2] * tmp_pt[0] + cam2base[1 * 4 + 2] * tmp_pt[1] + cam2base[2 * 4 + 2] * tmp_pt[2];

        if (pt_cam_z <= 0)
        continue;

        int pt_pix_x = roundf(cam_K[0 * 3 + 0] * (pt_cam_x / pt_cam_z) + cam_K[0 * 3 + 2]);
        int pt_pix_y = roundf(cam_K[1 * 3 + 1] * (pt_cam_y / pt_cam_z) + cam_K[1 * 3 + 2]);
        if (pt_pix_x < 0 || pt_pix_x >= im_width || pt_pix_y < 0 || pt_pix_y >= im_height)
        continue;

        float depth_val = depth_im[pt_pix_y * im_width + pt_pix_x];

        if (depth_val <= 0 || depth_val > 6)
        continue;

        float diff = depth_val - pt_cam_z;

        if (diff <= -trunc_margin)
        continue;

        // Integrate
        int volume_idx = pt_grid_z * voxel_grid_dim_y * voxel_grid_dim_x + pt_grid_y * voxel_grid_dim_x + pt_grid_x;
        float dist = fmin(1.0f, diff / trunc_margin);
        float weight_old = voxel_grid_weight[volume_idx];
        float weight_new = weight_old + 1.0f;
        voxel_grid_weight[volume_idx] = weight_new;
        voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;
    }
}


// Loads a binary file with depth data and generates a TSDF voxel volume (5m x 5m x 5m at 1cm resolution)
// Volume is aligned with respect to the camera coordinates of the first frame (a.k.a. base frame)
extern "C" void TSDF_Fusion(const char * data_folder, int frame_nums, const float*target_pos, const char *save_path) {
    std::string ply_save_path = save_path;
    std::string data_path = data_folder;
    // Location of camera intrinsic file
    // std::string cam_K_file = data_path + "/camera-intrinsics_tuyang_rgb.txt";
    std::string cam_K_file = data_path + "/camera-intrinsics.txt";
    std::string adjust_hand_eye_file = data_path + "/adjust_hand_eye.txt";

    // Location of folder containing RGB-D frames and camera pose files
    int base_frame_idx = 0;
    int first_frame_idx = 0;
    float num_frames = frame_nums;

    float cam_K[3 * 3];
    float cam2tmp[4 * 4];
    float tmp2world[4 * 4];
    float base2world[4 * 4];
    float cam2base[4 * 4];
    float cam2world[4 * 4];
    int im_width = 640;
    int im_height = 480;
    float depth_im[im_height * im_width];
    //[-0.145, 0.414, 1.162]
    float world_voxel_grid_origin_x = -0.145f; // Location of voxel grid origin in base frame camera coordinates
    float world_voxel_grid_origin_y = 0.414;
    float world_voxel_grid_origin_z = 1.15;

    // Voxel grid parameters (change these to change voxel grid resolution, etc.)
    float voxel_grid_origin_x = -0.5f; // Location of voxel grid origin in base frame camera coordinates
    float voxel_grid_origin_y = -0.5f;
    float voxel_grid_origin_z = 0.3f;
    float voxel_size = 0.1f;
    // float voxel_size = 0.0005f;
    float trunc_margin = voxel_size * 5;
    int voxel_grid_dim_x = 500;
    int voxel_grid_dim_y = 500;
    int voxel_grid_dim_z = 500;

    // Read camera intrinsics
    std::cout << "Read camera intrinsics\n";
    std::vector<float> cam_K_vec = LoadMatrixFromFile(cam_K_file, 3, 3);
    std::copy(cam_K_vec.begin(), cam_K_vec.end(), cam_K);

    // Read base frame camera pose
    std::cout << "Read base frame camera pose\n";
    std::ostringstream base_frame_prefix;
    base_frame_prefix << std::setw(2) << std::setfill('0') << base_frame_idx;
    std::string base2world_file = data_path + "/frame_" + base_frame_prefix.str() + "_pose.txt";
    std::vector<float> tmp2world_vec = LoadMatrixFromFile(base2world_file, 4, 4);
    std::copy(tmp2world_vec.begin(), tmp2world_vec.end(), tmp2world);
    std::vector<float> cam2tmp_vec = LoadMatrixFromFile(adjust_hand_eye_file, 4, 4);
    std::copy(cam2tmp_vec.begin(), cam2tmp_vec.end(), cam2tmp);
    multiply_matrix(tmp2world, cam2tmp, base2world);

    // Invert base frame camera pose to get world-to-base frame transform 
    float base2world_inv[16] = {0};
    invert_matrix(base2world, base2world_inv);

    float in_pt[3] = {world_voxel_grid_origin_x, world_voxel_grid_origin_y, world_voxel_grid_origin_z};
    float out_pt[3] = {0};
    transform_point(base2world_inv, in_pt, out_pt);
    voxel_grid_origin_x = out_pt[0] - voxel_grid_dim_x*voxel_size/2;
    voxel_grid_origin_y = out_pt[1] - voxel_grid_dim_y*voxel_size/2;
    voxel_grid_origin_z = out_pt[2] - voxel_grid_dim_z*voxel_size/2;
    std::cout <<"voxel_grid_origin: ";
    std::cout <<voxel_grid_origin_x <<","<<voxel_grid_origin_y <<","<<voxel_grid_origin_z <<"\n";

    // Initialize voxel grid
    std::cout << "Initialize voxel grid\n";
    float * voxel_grid_TSDF = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
    float * voxel_grid_weight = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
    for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
        voxel_grid_TSDF[i] = 1.0f;
    memset(voxel_grid_weight, 0, sizeof(float) * voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z);

    // Load variables to GPU memory
    float * gpu_voxel_grid_TSDF;
    float * gpu_voxel_grid_weight;
    hipMalloc(&gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
    hipMalloc(&gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
    checkCUDA(__LINE__, hipGetLastError());
    hipMemcpy(gpu_voxel_grid_TSDF, voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_voxel_grid_weight, voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyHostToDevice);
    checkCUDA(__LINE__, hipGetLastError());
    float * gpu_cam_K;
    float * gpu_cam2base;
    float * gpu_depth_im;
    hipMalloc(&gpu_cam_K, 3 * 3 * sizeof(float));
    hipMemcpy(gpu_cam_K, cam_K, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&gpu_cam2base, 4 * 4 * sizeof(float));
    hipMalloc(&gpu_depth_im, im_height * im_width * sizeof(float));
    checkCUDA(__LINE__, hipGetLastError());

    // Loop through each depth frame and integrate TSDF voxel grid
    for (int frame_idx = first_frame_idx; frame_idx < first_frame_idx + (int)num_frames; ++frame_idx) {

        std::ostringstream curr_frame_prefix;
        curr_frame_prefix << std::setw(2) << std::setfill('0') << frame_idx;

        // // Read current frame depth
        std::string depth_im_file = data_path + "/frame_" + curr_frame_prefix.str() + "_depth.png";
        std::cout << "Read current frame dept: " << depth_im_file << std::endl;
        ReadDepth(depth_im_file, im_height, im_width, depth_im);

        // Read base frame camera pose
        // std::string cam2world_file = data_path + "/frame_" + curr_frame_prefix.str() + "_pose.txt";
        // std::cout << "Read base frame camera pose: " << cam2world_file << std::endl;
        // std::vector<float> cam2world_vec = LoadMatrixFromFile(cam2world_file, 4, 4);
        // for (float n: cam2world_vec)
        //   std::cout<< n <<std::endl;
        // std::copy(cam2world_vec.begin(), cam2world_vec.end(), cam2world);
        std::string cam2world_file = data_path + "/frame_" + curr_frame_prefix.str() + "_pose.txt";
        std::cout << "Read base frame camera pose: " << cam2world_file << std::endl;
        std::vector<float> tmp2world_vec = LoadMatrixFromFile(cam2world_file, 4, 4);
        std::copy(tmp2world_vec.begin(), tmp2world_vec.end(), tmp2world);
        multiply_matrix(tmp2world, cam2tmp, cam2world);

        // Compute relative camera pose (camera-to-base frame)
        multiply_matrix(base2world_inv, cam2world, cam2base);

        hipMemcpy(gpu_cam2base, cam2base, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(gpu_depth_im, depth_im, im_height * im_width * sizeof(float), hipMemcpyHostToDevice);
        checkCUDA(__LINE__, hipGetLastError());

        std::cout << "Fusing: " << depth_im_file << std::endl;

        Integrate <<< voxel_grid_dim_z, voxel_grid_dim_y >>>(gpu_cam_K, gpu_cam2base, gpu_depth_im,
                                                            im_height, im_width, voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z,
                                                            voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z, voxel_size, trunc_margin,
                                                            gpu_voxel_grid_TSDF, gpu_voxel_grid_weight);
    }

    // Load TSDF voxel grid from GPU to CPU memory
    hipMemcpy(voxel_grid_TSDF, gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(voxel_grid_weight, gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyDeviceToHost);
    checkCUDA(__LINE__, hipGetLastError());

    hipFree(gpu_voxel_grid_TSDF);
    hipFree(gpu_voxel_grid_weight);
    hipFree(gpu_cam_K);
    hipFree(gpu_cam2base);
    hipFree(gpu_depth_im);

    // Compute surface points from TSDF voxel grid and save to point cloud .ply file
    std::cout << "Saving surface point cloud : " << ply_save_path << std::endl;
    SaveVoxelGrid2SurfacePointCloud(ply_save_path, voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z, 
                                    voxel_size, voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z,
                                    voxel_grid_TSDF, voxel_grid_weight, 0.2f, 0.0f, base2world);
}


